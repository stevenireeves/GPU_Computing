#include <stdio.h>
// Query code, mostly borrowed from the internets.
// Print device properties
void printDevProp(hipDeviceProp_t devProp) {
  printf("Name:                          %s\n", devProp.name);
  printf("Total global memory:           %zu\n", devProp.totalGlobalMem);
  printf("Total shared memory per block: %zu\n", devProp.sharedMemPerBlock);
  printf("Total registers per block:     %d\n", devProp.regsPerBlock);
  printf("Warp size:                     %d\n", devProp.warpSize);
  printf("Maximum memory pitch:          %zu\n", devProp.memPitch);
  printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
  for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
  for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
  printf("Clock rate:                    %d\n", devProp.clockRate);
  printf("Total constant memory:         %zu\n", devProp.totalConstMem);
  printf("Texture alignment:             %zu\n", devProp.textureAlignment);
  printf("Concurrent copy and execution: %s\n",
         (devProp.deviceOverlap ? "Yes" : "No"));
  printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
  printf("Kernel execution timeout:      %s\n",
         (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
  return;
}

int main() {
  // Number of CUDA devices
  int devCount;
  hipGetDeviceCount(&devCount);
  printf("CUDA Device Query...\n");
  printf("There are %d CUDA devices.\n", devCount);

  // Iterate through devices
  for (int i = 0; i < devCount; ++i) {
    // Get device properties
    printf("\nCUDA Device #%d\n", i);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, i);
    printDevProp(devProp);
  }

  printf("\nPress any key to exit...");
  char c;
  scanf("%c", &c);

  return 0;
}
