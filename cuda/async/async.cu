/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(float *a, int offset) {
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  float x = (float)i;
  float s = sinf(x);
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s * s + c * c);
}

float maxError(float *a, int n) {
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i] - 1.0f);
    if (error > maxE)
      maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv) {
  const int blockSize = 256, nStreams = 4;
  const int n = 4 * 1024 * blockSize * nStreams;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);

  int devId = 0;
  if (argc > 1)
    devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda(hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda(hipSetDevice(devId));

  // allocate pinned host memory and device memory
  float *a, *d_a;
  checkCuda(hipHostMalloc((void **)&a, bytes, hipHostMallocDefault)); // host pinned
  checkCuda(hipMalloc((void **)&d_a, bytes));   // device

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  checkCuda(hipEventCreate(&dummyEvent));
  for (int i = 0; i < nStreams; ++i)
    checkCuda(hipStreamCreate(&stream[i]));

  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);
  checkCuda(hipEventRecord(startEvent, 0));
  checkCuda(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
  kernel<<<n / blockSize, blockSize>>>(d_a, 0);
  checkCuda(hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("Time for sequential transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda(hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes,
                              hipMemcpyHostToDevice, stream[i]));
    kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    checkCuda(hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes,
                              hipMemcpyDeviceToHost, stream[i]));
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 2:
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda(hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes,
                              hipMemcpyHostToDevice, stream[i]));
  }
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
  }
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda(hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes,
                              hipMemcpyDeviceToHost, stream[i]));
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // cleanup
  checkCuda(hipEventDestroy(startEvent));
  checkCuda(hipEventDestroy(stopEvent));
  checkCuda(hipEventDestroy(dummyEvent));
  for (int i = 0; i < nStreams; ++i)
    checkCuda(hipStreamDestroy(stream[i]));
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}
